#include<hip/hip_runtime_api.h>
#include"timeInterval.h"

// this class is intended to compute the time spent between two reference points.
// the computation is started by calling the following method
void Interval::startInterval(){
	hipEventRecord (start, 0);
}

// the following method finishs the timer and computes the interval length
void Interval::endInterval(){
	hipEventRecord (end,   0);
	hipEventSynchronize ( end );
	hipEventElapsedTime ( &elapsedTime, start, end);
}

// return the interval length
float Interval::getTime(){
	return elapsedTime/1000;
}

// constructor , initializing variables
Interval::Interval(){
	hipEventCreate( &start );
	hipEventCreate( &end   );
	elapsedTime = 0;
}

// cleaning variables from memory
Interval::~Interval(){
	hipEventDestroy( start );
	hipEventDestroy( end   );
}
